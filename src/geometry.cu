#include "hip/hip_runtime.h"
#include <tsx/prefix.h>

namespace	tsx{

	Rectangle::Rectangle()
	: w(0), h(0){
		limits.max.width	= null;
		limits.max.height	= null;
		limits.min.width	= null;
		limits.min.height	= null;
	}

	Rectangle::Rectangle(uint W, uint H)
	: w(W), h(H){
		limits.max.width	= null;
		limits.max.height	= null;
		limits.min.width	= null;
		limits.min.height	= null;
	}

	Rectangle::Rectangle(const Rectangle & rect)
	: w(rect.w), h(rect.h){
		
	}

	Rectangle::~Rectangle(){
		if( limits.max.width isnot null )
			delete	limits.max.width;

		if( limits.max.height isnot null )
			delete	limits.max.height;

		if( limits.min.width isnot null )
			delete	limits.min.width;

		if( limits.min.height isnot null )
			delete	limits.min.height;

		limits.max.width = limits.max.height = null;
		limits.min.width = limits.min.height = null;
	}

}


