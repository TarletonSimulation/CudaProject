#include "hip/hip_runtime.h"
#include <tsx/prefix.h>
#include <tsx/geometry.h>

namespace	tsx{

	Rectangle::Rectangle()
	: w(0), h(0){
		w_lock = false;
		h_lock = false;
	}

	Rectangle::Rectangle(float W, float H)
	: w(W), h(H){
		w_lock = false;
		h_lock = false;
	}

	Rectangle::Rectangle(const Rectangle & rect)
	: w(rect.w), h(rect.h){
		w_lock = rect.w_lock;
		h_lock = rect.h_lock;
	}

	Rectangle::~Rectangle(){
		// just in case the object holds an address for whatever reason //
		w_lock  = false;
		h_lock  = false;
	}

	// friend functions //

	void
	set( Rectangle & A, float W, float H ){
		if( A.width_locked() is false )
			A.width( W );
		if( A.height_locked() is false )
			A.height( H );
	}

	void
	Rectangle::set( Rectangle & A, float W, float H ){
		tsx::set(A,W,H);
	}

	void
	set( Rectangle & A, const Rectangle & B ){
		set(A,B.width(),B.height());
		
		A.lock_width( B.width_locked() );
		A.lock_height( B.height_locked() );
	}

	void
	Rectangle::set( Rectangle & A, const Rectangle & B ){
		tsx::set(A,B);
	}

	const Rectangle &
	add( const Rectangle & A, const Rectangle & B ){
		Rectangle C( A.width() + B.width(), A.height() + B.height() );

		if( (A.width_locked() is true) or (B.width_locked() is true) )
			C.lock_width(true);
		else	C.lock_width(false);

		if( (A.height_locked() is true) or (B.height_locked() is true) )
			C.lock_height(true);
		else	C.lock_height(false);
	}

	const Rectangle &
	Rectangle::add(const Rectangle & A, const Rectangle & B){
		return	tsx::add(A,B);
	}

	const Rectangle &
	Rectangle::add(const Rectangle & A)
	const{
		return	tsx::add(*this,A);
	}

	const Rectangle &
	add(const Rectangle & A, float a, float b){
		Rectangle C(A);
		C.width( C.width() + a );
		C.height( C.height() + b );
	return	C;
	}

	const Rectangle &
	Rectangle::add(const Rectangle & A, float a, float b){
		return	tsx::add(A,a,b);
	}

	const Rectangle &
	Rectangle::add(float a, float b)
	const{
		return	tsx::add(*this,a,b);
	}

	const Rectangle &
	add_to( Rectangle & A, const Rectangle & B ){
		if( A.width_locked() is false )
			A.width( A.width() + B.width() );
		if( A.height_locked() is false )
			A.height( A.height() + B.height() );
	return	A;
	}

	const Rectangle &
	Rectangle::add_to( Rectangle & A, const Rectangle & B ){
		return	tsx::add_to(A,B);
	}

	const Rectangle &
	sub(const Rectangle & A, const Rectangle & B){
		Rectangle C( A.width() - B.width(), A.height() - B.height() );

		if( (A.width_locked() is true) or (B.width_locked() is true) )
			C.lock_width(true);
		else	C.lock_width(false);

		if( (A.height_locked() is true) or (B.height_locked() is true) )
			C.lock_height(true);
		else	C.lock_height(false);
	}

	const Rectangle &
	Rectangle::sub( const Rectangle & A, const Rectangle & B ){
		return	tsx::sub(A,B);
	}

	const Rectangle &
	sub(const Rectangle & A, float W, float H){
		Rectangle C( A.width() - W, A.height() - H );
		
		if( A.width_locked() is true )
			C.lock_width();
		if( A.height_locked() is true )
			C.lock_height();

	return	C;
	}

	const Rectangle &
	Rectangle::sub(const Rectangle & A, float W, float H){
		return	tsx::sub(A,W,H);
	}

	const Rectangle &
	Rectangle::sub(float W, float H)
	const{
		return	tsx::sub(*this,W,H);
	}

	const Rectangle &
	sub_from(Rectangle & A, const Rectangle & B){
		if( A.width_locked() is false )
			A.width( A.width() - B.width() );
		if( A.height_locked() is false )
			A.height( A.height() - B.height() );
	return	A;
	}

	const Rectangle &
	Rectangle::sub_from(Rectangle & A, const Rectangle & B){
		return	tsx::sub_from(A,B);
	}
	
	const Rectangle &
	scale(Rectangle & A, float x){
		A.width( (float)(A.width()*x) );
		A.height( (float)(A.height()*x) );
	return	A;
	}

	const Rectangle &
	Rectangle::scale(Rectangle & A, float X){
		return	tsx::scale(A,X);
	}

	const Rectangle &
	scale(Rectangle & A, float x, float y){
		A.width( A.width()*x );
		A.height( A.height()*y );
	return	A;
	}

	const Rectangle &
	Rectangle::scale(Rectangle & A, float x, float y){
		return	tsx::scale(A,x,y);
	}


	Rectangle *
	free_rectangle(Rectangle * rect){
		if( rect is null )
			return	rect;
		else	delete	rect;

	return	(rect = null);
	}

	Rectangle *
	Rectangle::free_rectangle(Rectangle * rect){
		return	tsx::free_rectangle(rect);
	}

	// end statics and friend functions //

	Rectangle
	Rectangle::create(float W, float H){
		return	Rectangle(W,H);
	}

	Rectangle *
	Rectangle::create_address(float W, float H){
		return	new Rectangle(W,H);
	}

	float
	Rectangle::area(const Rectangle & rect){
		return	rect.area();
	}

	float
	Rectangle::width(const Rectangle & rect){
		return	rect.width();
	}

	float
	Rectangle::height(const Rectangle & rect){
		return	rect.height();
	}

	bool
	Rectangle::width_locked(const Rectangle & rect){
		return	rect.width_locked();
	}

	bool
	Rectangle::height_locked(const Rectangle & rect){
		return	rect.height_locked();
	}

	// end static methods //
	
	void
	Rectangle::remove_locks(){
		w_lock = false;
		h_lock = false;
	}

	void
	Rectangle::width(float W){
		if( w_lock is true )
			return;
		else	w = W;
	}

	float
	Rectangle::width()
	const{return	w;}

	void
	Rectangle::lock_width(bool lock){
		w_lock = lock;
	}

	void
	Rectangle::height(float H){
		if( h_lock is true )
			return;
		else	h = H;
	}

	void
	Rectangle::lock_height(bool lock){
		h_lock = lock;
	}

	bool
	Rectangle::height_locked()
	const{return	(h_lock is true);}

	bool
	Rectangle::width_locked()
	const{return	(w_lock is true);}

	float
	Rectangle::height()
	const{return	h;}

	void
	Rectangle::rectangle(float W, float H){
		if( w_lock is false )
			w = W;
		if( h_lock is false )
			h = H;
	}

	void
	Rectangle::rectangle(const Rectangle & rect){
		rectangle(rect.w, rect.h);

		w_lock = rect.w_lock;
		h_lock = rect.h_lock;
	}

	const Rectangle &
	Rectangle::rectangle()
	const{return	*this;}

	Rectangle *
	Rectangle::rectangle_address()
	{return	this;}

	Rectangle &
	Rectangle::rectangle_ref()
	{return	*this;}

	float
	Rectangle::area()
	const {return	width()*height();}

	float
	Rectangle::perimeter()
	const{return 2*(w+h);}

	float
	Rectangle::magnitude()
	const{
		return	sqrtf( area() );
	}



	bool
	Rectangle::operator	== (const Rectangle & rect){
		if( w isnot rect.w )
			return	false;
	else	if( h isnot rect.h )
			return	false;
	else	if( w_lock isnot rect.w_lock )
			return	false;
	else	if( h_lock isnot rect.h_lock )
			return	false;
		else	return	true;
	}

	bool
	Rectangle::operator	!= (const Rectangle & rect){
		return	( (*this is rect) is false );
	}

	const Rectangle &
	Rectangle::operator	=  (const Rectangle & rect){
		tsx::set(*this,rect);
	return	*this;
	}

	const Rectangle &
	Rectangle::operator	+ (const Rectangle & rect){
		return tsx::add(*this,rect);
	}

	const Rectangle &
	Rectangle::operator	+= (const Rectangle & rect){
		return	tsx::add_to(*this,rect);
	}

	const Rectangle &
	Rectangle::operator	-  (const Rectangle & rect){
		return	tsx::sub(*this,rect);
	}

	const Rectangle &
	Rectangle::operator	-= (const Rectangle & rect){
		return	tsx::sub_from(*this,rect);
	}

	const Rectangle &
	Rectangle::operator	*  ( float x ){
		tsx::scale(*this,x);
	}

	// private methods //
	
	bool
	Point::can_set_x()
	const{
		if( point_locked() is true )
			return	false;
	else	if( (ly is true) or (lz is true) )
			return	false;
		else	return	true;
	}

	bool
	Point::can_set_y()
	const{
		if( point_locked() is true )
			return	false;
	else	if( (lx is true) or (lz is true) )
			return	false;
		else	return	true;
	}

	bool
	Point::can_set_z()
	const{
		if( point_locked() is true )
			return	false;
	else	if( (ly is true) or (lx is true) )
			return	false;
		else	return	true;
	}

	// end private //


	Point::Point()
	: px(0.0f), py(0.0f), pz(0.0f){
		lx = ly = lz = false;
	}

	Point::Point(float X, float Y, float Z)
	: px(X), py(Y), pz(Z){
		lx = ly = lz = false;
	}

	Point::Point(const Point & A){
		px = A.x();
		py = A.y();
		pz = A.z();

		lx = A.x_locked();
		ly = A.y_locked();
		lz = A.z_locked();
	}

	Point::~Point(){}

	const Point &
	Point::point()
	const{return *this;}

	Point *
	Point::point_address()
	{return this;}

	Point &
	Point::point_ref()
	{return *this;}

	// statics and friend zone //

	Point
	Point::create(float a, float b, float c){
		return	Point(a,b,c);
	}

	Point *
	Point::create_address(float a, float b, float c){
		return	new Point(a,b,c);
	}

	Point
	add(const Point & a, const Point & b){
		Point c( a.x() + b.x(), a.y() + b.y(), a.z() + b.z() );

		if( (a.x_locked() is true) or (b.x_locked() is true) )
			c.lock_x();

		if( (a.y_locked() is true) or (b.y_locked() is true) )
			c.lock_y();

		if( (a.z_locked() is true) or (b.z_locked() is true) )
			c.lock_z();

	return	c;
	}

	Point
	Point::add(const Point & a, const Point & b){
		return	tsx::add(a,b);
	}

	Point
	Point::add(const Point & b)
	const{
		return	tsx::add(*this,b);
	}

	Point
	add(const Point & A, float a, float b, float c){
		Point u(A);

		u.x( u.x() + a );
		u.y( u.y() + b );
		u.z( u.z() + c );
	return	u;
	}

	Point
	Point::add(const Point & A, float a, float b, float c){
		return	tsx::add(A,a,b,c);
	}

	Point
	Point::add(float a, float b, float c)
	const{
		return	tsx::add(*this,a,b,c);
	}

	const Point &
	add_to(Point & a, const Point & b){
		a.x( a.x() + b.x() );
		a.y( a.y() + b.y() );
		a.z( a.z() + b.z() );
	return	a;
	}

	const Point &
	Point::add_to(Point & a, const Point & b){
		return	tsx::add_to(a,b);
	}

	const Point &
	scale(Point & a, float all){
		a.x( a.x()*all );
		a.y( a.y()*all );
		a.z( a.z()*all );
	return	a;
	}

	const Point &
	Point::scale(Point & a, float all){
		return	tsx::scale(a,all);
	}

	const Point &
	Point::scale(float all){
		return	tsx::scale(*this,all);
	}

	const Point &
	scale(Point & a, float e1, float e2, std::string axiis){
		if( axiis is "xy" ){
			a.x( a.x()*e1 );
			a.y( a.y()*e2 );
	}else	if( axiis is "yz" ){
			a.y( a.y()*e1 );
			a.z( a.z()*e2 );
	}else	if( axiis is "zx" ){
			a.z( a.z()*e1 );
			a.x( a.x()*e2 );
	}else	if( axiis is "yx" ){
			a.y( a.y()*e1 );
			a.x( a.x()*e2 );
	}else	if( axiis is "zy" ){
			a.z( a.z()*e1 );
			a.y( a.y()*e2 );
	}else	if( axiis is "xz" ){
			a.x( a.x()*e1 );
			a.z( a.z()*e2 );
	}

	return	a;
	}

	const Point &
	Point::scale(Point & a, float e1, float e2, std::string axiis){
		return	tsx::scale(a,e1,e2,axiis);
	}

	const Point &
	Point::scale(float a, float b, std::string aa){
		return	tsx::scale(*this,a,b,aa);
	}

	const Point &
	scale(Point & a, float w, float u, float v){
		a.x( a.x()*w );
		a.y( a.y()*u );
		a.z( a.z()*v );
	return	a;
	}

	const Point &
	Point::scale(Point & a, float w, float u, float v){
		return	tsx::scale(a,w,u,v);
	}

	const Point &
	Point::scale(float w, float u, float v){
		return	tsx::scale(*this,w,u,v);
	}

	const Point &
	scale(Point & a, const Point & b){
		return	tsx::scale(a,b.x(), b.y(), b.z());
	}

	const Point &
	Point::scale(Point & a, const Point & b){
		return	tsx::scale(a,b);
	}

	const Point &
	Point::scale(const Point & b){
		return	tsx::scale(*this,b);
	}

	float
	product(const Point & a, const Point & b){
		float sum = 0.0f;
		sum += a.x() * b.x();
		sum += a.y() * b.y();
		sum += a.z() * b.z();
	return	sum;
	}

	float
	Point::product(const Point & a, const Point & b){
		return	tsx::product(a,b);
	}

	float
	Point::product(const Point & a)
	const{
		return	tsx::product(*this,a);
	}

	void
	test_func()
	{std::cout << "Testing" << std::endl;}

	float
	distance(const Point & A, const Point & B){
		float	dx = A.x() - B.x();
		float	dy = A.y() - B.y();
		float	dz = A.z() - B.z();

		float	sx = dx*dx;
		float	sy = dy*dy;
		float	sz = dz*dz;
	return	sqrtf(sx+sy+sz);
	}

	float
	Point::distance(const Point & A, const Point & B){
		return	tsx::distance(A,B);
	}

	float
	Point::distance(const Point & A)
	const{
		return	distance(*this,A);
	}

	float
	distance(const Point & A, float u, float v, float w){
		float	dx = A.x() - u;
		float	dy = A.y() - v;
		float	dz = A.z() - w;

		float	sx = dx*dx;
		float	sy = dy*dy;
		float	sz = dz*dz;
	return	sqrtf(sx+sy+sz);
	}

	float
	Point::distance(const Point & A, float u, float v, float w){
		return	tsx::distance(A,u,v,w);
	}

	float
	Point::distance(float u, float v, float w)
	const{
		return	tsx::distance(*this,u,v,w);
	}

	float
	magnitude(const Point & A){
		return	tsx::distance(A,0.0f,0.0f,0.0f);
	}

	float
	Point::magnitude(const Point & A){
		return	tsx::magnitude(A);
	}

	float
	Point::magnitude()
	const{return	magnitude(*this);}

	// end friends and statics //
	
	bool
	Point::point_locked()
	const{return ( (lx is true) and (ly is true) and (lz is true) );}

	void
	Point::lock_point(bool lock){
		lx = ly = lz = lock;
	}

	float
	Point::x()
	const{return px;}

	void
	Point::x(float nx){
		if( can_set_x() is true )
			px = nx;
	}

	void
	Point::y(float ny){
		if( can_set_y() is true )
			py = ny;
	}

	void
	Point::z(float nz){
		if( can_set_z() is true )
			pz = nz;
	}

	float
	Point::y()
	const{return py;}

	float
	Point::z()
	const{return pz;}

	bool
	Point::x_locked()
	const{return lx;}

	bool
	Point::y_locked()
	const{return ly;}

	bool
	Point::z_locked()
	const{return lz;}


	void
	Point::lock_x(bool lock)
	{lx=lock;}

	void
	Point::lock_y(bool lock)
	{ly=lock;}

	void
	Point::lock_z(bool lock)
	{lz=lock;}

	void
	Point::remove_locks(){
		lx = ly = lz = false;
	}




}



