#include "hip/hip_runtime.h"
#include <tsx/geometry.h>

namespace	tsx{

	Rectangle::Rectangle()
	: w(0), h(0){
		max = nullptr;
		min = nullptr;
		iter= nullptr;

		iter_shared	= false;
		max_shared	= false;
		min_shared	= false;
	}

	Rectangle::Rectangle(uint W, uint H)
	: w(W), h(H){
		max = nullptr;
		min = nullptr;
		iter= nullptr;

		iter_shared	= false;
		max_shared	= false;
		min_shared	= false;
	}

	Rectangle::Rectangle(const Rectangle & rect)
	: w(rect.w), h(rect.h){
		if( rect.max isnot null )	init_max(rect.max->w, rect.max->h);
		if( rect.min isnot null )	init_min(rect.min->w, rect.min->h);
		if( rect.iter isnot null )	iteration(rect.iter->w, rect.iter->h);
	}

	Rectangle::~Rectangle(){
		if( min isnot null )
			delete min;
		if( max isnot null )
			delete max;
		if( (iter isnot null) and (iter_shared is false) )
			delete iter;
		else	if( iter_shared is true )
			iter = null;

		iter	= null;
		max	= null;
		min	= null;
	}

	// friend methods //
	
	Rectangle
	add( const Rectangle & A, const Rectangle & B){
		Rectangle C(A);

		C.w += B.width();
		C.h += B.height();

		C.remove_limits();
		C.remove_iterator();
	return	C;
	}

	const Rectangle &
	add_to(Rectangle & A, const Rectangle & B){
		A.w += B.width();
		A.h += B.height();
	return	A;
	}

	// end friends //

	// static methods //

	Rectangle
	Rectangle::create(uint w, uint h){
		Rectangle A(w,h);
	return	A;
	}

	Rectangle *
	Rectangle::create_pointer(uint w, uint h){
		return	new Rectangle(w,h);
	}

	// end statics //

	// protected methods //

	bool
	Rectangle::has_limits()
	const{
		if( (has_max() is true) or (has_min() is true) )
			return	true;
		else	return	false;
	}

	bool
	Rectangle::has_max()
	const{
		if( max is null )
			return	false;
		else	return	true;
	}

	bool
	Rectangle::has_min()
	const{
		if( min is null )
			return	false;
		else	return	true;
	}

	bool
	Rectangle::has_iterator()
	const{
		if( iter is null )
			return	false;
		else	return	true;
	}

	void
	Rectangle::init_max(uint mw, uint mh){
		if( has_max() ){
			max_rectangle(mw,mh);
			return;
		}

		max = new Rectangle(mw,mh);
		return;
	}

	void
	Rectangle::init_min(uint mw, uint mh){
		if( has_min() ){
			min_rectangle(mw, mh);
			return;
		}

		min = new Rectangle(mw,mh);
		return;
	}

	void
	Rectangle::no_double_iter(){
		if( has_iterator() is false )
			return;

		if( iter->iter isnot null ){
			if( iter->iter_shared is true )
				iter->iter = null;
			else	delete	iter->iter;
		}
	}

	void
	Rectangle::no_double_limits(){
		if( has_max() is true ){
			if( max->max isnot null ){
				if( max->max_shared is true )
					max->max = null;
				else	delete	max->max;
			}
		}

		if( has_min() is true ){
			if( min->min isnot null ){
				if( min->min_shared is true )
					min->min = null;
				else	delete	min->min;
			}
		}
	}

	// end protected //

	void
	Rectangle::share_iteration(Rectangle * rect){
		if( rect is null )
			return;
			
		iter = rect->iter;
		iter_shared = true;

		no_double_iter();
	}

	bool
	Rectangle::shared_iterator() const
	{return	(iter_shared is true);}

	void
	Rectangle::share_max(Rectangle * rect){
		if( rect is null )
			return;
	else	if( rect->has_max() is false )
			return;
		else{
			if( ( max_shared is false ) and ( has_max() is true ) ){
				remove_max();
			}
				
			max = rect->max;
			max_shared = true;
		}
	}

	bool
	Rectangle::shared_max() const
	{return	(max_shared is true);}

	bool
	Rectangle::shared_min() const
	{return	(min_shared is true);}

	bool
	Rectangle::shared_limits() const
	{return	( (shared_min() is true) and (shared_max() is true) );}

	void
	Rectangle::share_min(Rectangle * rect){
		if( rect is null )
			return;
	else	if( rect->has_min() is false )
			return;
		else{
			if( ( min_shared is false ) and ( has_min() is true ) ){
				remove_min();
			}

			min = rect->min;
			min_shared = true;
		}
	}

	void
	Rectangle::remove_iterator(){
		if( iter is null )
			return;

		no_double_iter();

		if( iter_shared is true )
			iter = null;
		else	delete iter;

		iter = null;
	}

	void
	Rectangle::remove_max(){
		if(has_max() is false)
			return;
		
		no_double_limits();
		delete	max;
		max = nullptr;
	}

	void
	Rectangle::remove_min(){
		if( has_min() is false )
			return;

		no_double_limits();
		delete	min;
		min = nullptr;
	}

	void
	Rectangle::remove_limits(){
		if( has_limits() is false )
			return;
		remove_min();
		remove_max();
	}

	void
	Rectangle::iteration(int wi, int hi){
		if( iter isnot null ){
			iter->w = wi;
			iter->h = hi;
		return;
		}

		iter = new Rectangle(wi,hi);
	return;
	}

	void
	Rectangle::iteration(const Rectangle & rect, bool keep){
		if( keep is false ){
			remove_iterator();
			iter = new Rectangle(rect);

		}else	*iter = rect;

		no_double_iter();
	}

	Rectangle *
	Rectangle::iterator(){
		if( iter is null )
			return	null;	// return raw null pointer so the address isn't saved outside this class //
					// it must be set before it can be manipulated outside it's class //
					// for destructor and sharing reasons //
		return	iter;
	}

	void
	Rectangle::width(uint W){
		if( has_min() ){
			if( has_max() ){
				if( W gt max->w ){
					w = max->w;
					return;
				}
			}

			if( W lt min->w )
				w = min->w;
			else	w = W;
		return;
		}else	w = W;
	return;
	}

	void
	Rectangle::height(uint H){
		if( has_min() ){
			if( has_max() ){
				if( H gt max->h ){
					h = max->h;
					return;
				}
			}

			if( H lt min->h )
				h = min->h;
			else	h = H;
		return;
		}else	h = H;
	return;
	}

	uint
	Rectangle::width()
	const{return	w;}

	uint
	Rectangle::height()
	const{return	h;}

	void
	Rectangle::max_rectangle(uint mw, uint mh){
		if( has_max() ){
			if( has_min() ){
				if( *min is *max )
					return;
			}

			max->w = mw;
			max->h = mh;
		}else	init_max(mw,mh);

		if( max->max isnot null ){
			if( max->max_shared is true )
				max->max = null;
			else	delete	max->max;

			max->max = null;
		}

	return;
	}

	void
	Rectangle::min_rectangle(uint mw, uint mh){
		if( has_min() ){
			if( has_max() ){
				if( *min is *max )
					return;
			}

			min->w = mw;
			min->h = mh;
		}else	init_min(mw,mh);
		
		if( min->min isnot null ){
			if( min->min_shared is true )
				min->min = null;
			else	delete	min->min;

			min->min = null;
		}

	return;
	}

	const Rectangle &
	Rectangle::min_rectangle()
	const{
		if( has_min() )
			return	*min;
		else	return	create(0,0);
	}

	const Rectangle &
	Rectangle::max_rectangle()
	const{
		if( has_max() )
			return	*max;
		else	return	create(5000,5000);
	}

	Rectangle *
	Rectangle::rectangle_pointer()
	{return	this;}

	Rectangle *
	Rectangle::rectangle_max_pointer(){
		if( has_max() is false )
			return	null;
		else	return	max;
	}

	Rectangle *
	Rectangle::rectangle_min_pointer(){
		if( has_min() is false )
			return	null;
		else	return	min;
	}

	Rectangle &
	Rectangle::rectangle_ref()
	{return	*this;}

	Rectangle &
	Rectangle::rectangle_max_ref(){
		if( has_max() is false ){
			init_max();
		}return	*max;
	}

	Rectangle &
	Rectangle::rectangle_min_ref(){
		if( has_min() is false ){
			init_min();
		}return	*min;
	}



	bool
	Rectangle::operator == (const Rectangle & rect){
		if( w isnot rect.w )
			return	false;
	else	if( h isnot rect.h )
			return	false;
	else	if( has_max() isnot rect.has_max() )
			return	false;
	else	if( has_min() isnot rect.has_min() )
			return	false;
	else	if( has_iterator() isnot rect.has_iterator() )
			return	false;

		if( has_min() ){
			if( (min->w isnot rect.min->w) or (min->h isnot rect.min->h) )
				return	false;
		}

		if( has_max() ){
			if( (max->w isnot rect.max->w) or (max->h isnot rect.max->h) )
				return	false;
		}

		if( has_iterator() ){
			if( (iter->w isnot rect.iter->w) or (iter->h isnot rect.iter->h) )
				return	false;

			if( ( iter->has_limits() isnot rect.iter->has_limits() ) )
				return	false;

			if( iter->has_limits() ){
				if( (iter->has_max() isnot rect.iter->has_max()) )
					return	false;
			else	if( (iter->has_min() isnot rect.iter->has_min()) )
					return	false;
				
				if( iter->has_max() ){
					if(	(iter->max->w isnot rect.iter->max->w) or
						(iter->max->h isnot rect.iter->max->h) )
						return	false;
				}

				if( iter->has_min() ){
					if( 	(iter->min->w isnot rect.iter->min->w) or
						(iter->min->h isnot rect.iter->min->h) )
						return	false;
				}

			}
		}
	return	true;
	}

	bool
	Rectangle::operator != (const Rectangle & rect){
		return	( (*this == rect) is false );
	}


	const Rectangle &
	Rectangle::operator = (const Rectangle & rect){
		w = rect.w;
		h = rect.h;

		if( rect.has_max() is true ){
			if( has_max() is false )
				init_max( rect.max->w, rect.max->h );
		}

		if( rect.has_min() is true ){
			if( has_min() is false )
				init_min( rect.min->w, rect.min->h );
		}

		if( rect.has_iterator() ){
			iteration( *(rect.iter), true );
		}
	return	*this;
	}

	const Rectangle &
	Rectangle::operator + (const Rectangle & rect){
		return	add(*this,rect);
	}

	const Rectangle &
	Rectangle::operator += (const Rectangle & rect){
		return add_to(*this,rect);
	}

}
