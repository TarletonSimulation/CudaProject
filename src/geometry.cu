#include "hip/hip_runtime.h"
#include <tsx/prefix.h>
#include <tsx/geometry.h>

namespace	tsx{

	Rectangle::Rectangle()
	: w(0.0f), h(0.0f){
		w_lock = false;
		h_lock = false;

		auto_lock_set = false;
	}

	Rectangle::Rectangle(float W, float H)
	: w(W), h(H){
		w_lock = false;
		h_lock = false;

		auto_lock_set = false;
	}

	Rectangle::Rectangle(const Rectangle & rect){
		if( rect.w lte 0.0f )
			w = 1.0f;
		if( rect.h lte 0.0f )
			h = 1.0f;

		w_lock = rect.w_lock;
		h_lock = rect.h_lock;
		auto_lock_set = rect.auto_lock_set;
	}

	Rectangle::~Rectangle(){
		// just in case the object holds an address for whatever reason //
		w_lock  = false;
		h_lock  = false;
		auto_lock_set = false;
	}

	// friend functions //

	void
	auto_lock(Rectangle & rect, bool lock){
		rect.auto_lock_set = lock;
	}

	bool
	auto_locked(const Rectangle & rect){
		return	rect.auto_locked();
	}

	bool
	Rectangle::auto_locked(const Rectangle & rect){
		return	rect.auto_locked();
	}

	bool
	Rectangle::auto_locked()
	const{
		return	(auto_lock_set is true);
	}

	float
	safe_mult(float x){
		float y = 0.0f;
		if( can_use_scalar(x) is false )
			y = -x;
		else	y = x;
	return	y;
	}

	bool
	can_use_scalar(float x){
		if( x lte 0.0f )
			return	false;
		else	return	true;
	}

	float
	magnitude(const Rectangle & rect){
		return	rect.magnitude();
	}

	Rectangle
	copy_to_scale(const Rectangle & A, float x){
		if( can_use_scalar(x) is true )
			return	Rectangle( A.width()*x, A.height()*x );
		else	return	Rectangle( A );
	}

	Rectangle
	Rectangle::copy_to_scale(const Rectangle & A, float x){
		return	tsx::copy_to_scale(A,x);
	}

	Rectangle
	Rectangle::copy_to_scale(float x)
	const{
		return	tsx::copy_to_scale(*this,x);
	}

	Rectangle
	copy_to_scale(const Rectangle & A, float x, float y){
		float	u, v;
		if( can_use_scalar(x) is true )
			u = x;
		else	u = 1.0f;
		if( can_use_scalar(y) is true )
			v = y;
		else	v = 1.0f;
	return	Rectangle(A.width()*x, A.height()*y);
	}

	Rectangle
	Rectangle::copy_to_scale(const Rectangle & A, float x, float y){
		return	tsx::copy_to_scale(A,x,y);
	}

	Rectangle
	Rectangle::copy_to_scale(float x, float y)
	const{
		return	tsx::copy_to_scale(*this,x,y);
	}

	void
	set( Rectangle & A, float W, float H ){
		float x = safe_mult(W);
		float y = safe_mult(H);

		A.width(W);
		A.height(H);
	}

	void
	Rectangle::set( Rectangle & A, float W, float H ){
		tsx::set(A,W,H);
	}

	void
	Rectangle::set(float a, float b){
		tsx::set(*this,a,b);
	}

	void
	set( Rectangle & A, const Rectangle & B ){
		set(A,B.width(),B.height());
		
		if( A.auto_locked() is true ){
			A.lock_width( B.width_locked() );
			A.lock_height( B.height_locked() );
		}
	}

	void
	Rectangle::set( Rectangle & A, const Rectangle & B ){
		tsx::set(A,B);
	}

	void
	Rectangle::set(const Rectangle & A){
		tsx::set(*this,A);
	}

	Rectangle
	add( const Rectangle & A, const Rectangle & B ){
		Rectangle C( A.width() + B.width(), A.height() + B.height() );

		if( (A.width_locked() is true) or (B.width_locked() is true) )
			C.lock_width(true);
		else	C.lock_width(false);

		if( (A.height_locked() is true) or (B.height_locked() is true) )
			C.lock_height(true);
		else	C.lock_height(false);
	return	C;
	}

	Rectangle
	Rectangle::add(const Rectangle & A, const Rectangle & B){
		return	tsx::add(A,B);
	}

	const Rectangle &
	Rectangle::add(const Rectangle & A)
	const{
		return	tsx::add(*this,A);
	}

	Rectangle
	add(const Rectangle & A, float a, float b){
		Rectangle C(A);

		if( can_use_scalar(a) is false ){
			if( a isnot 0.0f ){
				if( (-1.0*a gte C.width()) is false )
					C.width( C.width() + a );
			}
		}

		if( can_use_scalar(b) is false ){
			if( b isnot 0.0f ){
				if( (-1.0*b gte C.height()) is false )
					C.height( C.height() + b );
			}
		}
	return	C;
	}

	Rectangle
	Rectangle::add(const Rectangle & A, float a, float b){
		return	tsx::add(A,a,b);
	}

	const Rectangle &
	Rectangle::add(float a, float b)
	const{
		return	tsx::add(*this,a,b);
	}

	const Rectangle &
	add_to( Rectangle & A, const Rectangle & B ){
		if( A.width_locked() is false )
			A.width( A.width() + B.width() );
		if( A.height_locked() is false )
			A.height( A.height() + B.height() );
	return	A;
	}

	const Rectangle &
	Rectangle::add_to( Rectangle & A, const Rectangle & B ){
		return	tsx::add_to(A,B);
	}

	Rectangle
	sub(const Rectangle & A, const Rectangle & B){
		Rectangle C( A.width() - B.width(), A.height() - B.height() );

		if( (A.width_locked() is true) or (B.width_locked() is true) )
			C.lock_width(true);
		else	C.lock_width(false);

		if( (A.height_locked() is true) or (B.height_locked() is true) )
			C.lock_height(true);
		else	C.lock_height(false);
	}

	Rectangle
	Rectangle::sub( const Rectangle & A, const Rectangle & B ){
		return	tsx::sub(A,B);
	}

	Rectangle
	sub(const Rectangle & A, float W, float H){
		Rectangle C( A.width() - W, A.height() - H );
		
		if( A.width_locked() is true )
			C.lock_width();
		if( A.height_locked() is true )
			C.lock_height();

	return	C;
	}

	Rectangle
	Rectangle::sub(const Rectangle & A, float W, float H){
		return	tsx::sub(A,W,H);
	}

	const Rectangle &
	Rectangle::sub(float W, float H)
	const{
		return	tsx::sub(*this,W,H);
	}

	const Rectangle &
	sub_from(Rectangle & A, const Rectangle & B){
		if( A.width_locked() is false )
			A.width( A.width() - B.width() );
		if( A.height_locked() is false )
			A.height( A.height() - B.height() );
	return	A;
	}

	const Rectangle &
	Rectangle::sub_from(Rectangle & A, const Rectangle & B){
		return	tsx::sub_from(A,B);
	}
	
	const Rectangle &
	scale(Rectangle & A, float x){
		A.width( (float)(A.width()*x) );
		A.height( (float)(A.height()*x) );
	return	A;
	}

	const Rectangle &
	Rectangle::scale(Rectangle & A, float X){
		return	tsx::scale(A,X);
	}

	const Rectangle &
	scale(Rectangle & A, float x, float y){
		A.width( A.width()*x );
		A.height( A.height()*y );
	return	A;
	}

	const Rectangle &
	Rectangle::scale(Rectangle & A, float x, float y){
		return	tsx::scale(A,x,y);
	}


	Rectangle *
	free_rectangle(Rectangle * rect){
		if( rect is null )
			return	rect;
		else	delete	rect;

	return	(rect = null);
	}

	Rectangle *
	Rectangle::free_rectangle(Rectangle * rect){
		return	tsx::free_rectangle(rect);
	}

	void
	lock(Rectangle & A, bool lock_w, bool lock_h){
		A.lock_width(lock_w);
		A.lock_height(lock_h);
	}

	void
	lock_width(Rectangle & A, bool lock){
		Rectangle::lock_width(A,lock);
	}

	void
	lock_height(Rectangle & A, bool lock){
		Rectangle::lock_height(A,lock);
	}

	void
	Rectangle::lock(Rectangle & A, bool x, bool y){
		tsx::lock(A,x,y);
	}

	void
	Rectangle::lock(bool x, bool y){
		tsx::lock(*this, x,y);
	}

	void
	Rectangle::lock_width(Rectangle & A, bool l){
		A.w_lock = l;
	}

	void
	Rectangle::lock_height(Rectangle & A, bool l){
		A.h_lock = l;
	}

	bool
	has_lock(const Rectangle & A){
		return	A.has_lock();
	}

	bool
	Rectangle::has_lock(const Rectangle & A){
		return	A.has_lock();
	}

	bool
	Rectangle::has_lock()
	const{
		return	( (w_lock is true) or (h_lock is true) );
	}

	// end statics and friend functions //

	Rectangle
	Rectangle::create(float W, float H){
		return	Rectangle(W,H);
	}

	Rectangle *
	Rectangle::create_address(float W, float H){
		return	new Rectangle(W,H);
	}

	float
	Rectangle::area(const Rectangle & rect){
		return	rect.area();
	}

	float
	Rectangle::width(const Rectangle & rect){
		return	rect.width();
	}

	float
	Rectangle::height(const Rectangle & rect){
		return	rect.height();
	}

	bool
	Rectangle::width_locked(const Rectangle & rect){
		return	rect.width_locked();
	}

	bool
	Rectangle::height_locked(const Rectangle & rect){
		return	rect.height_locked();
	}

	// end static methods //

	void
	remove_locks(Rectangle & a){
		a.w_lock = false;
		a.h_lock = false;
	}

	void
	Rectangle::remove_locks(Rectangle & a){
		tsx::remove_locks(a);
	}
	
	void
	Rectangle::remove_locks(){
		tsx::remove_locks(*this);
	}

	void
	Rectangle::width(float W){
		if( w_lock is true )
			return;
		else	w = W;
	}

	float
	Rectangle::width()
	const{return	w;}

	void
	Rectangle::lock_width(bool lock){
		w_lock = lock;
	}

	void
	Rectangle::height(float H){
		if( h_lock is true )
			return;
		else	h = H;
	}

	void
	Rectangle::lock_height(bool lock){
		h_lock = lock;
	}

	bool
	Rectangle::height_locked()
	const{return	(h_lock is true);}

	bool
	Rectangle::width_locked()
	const{return	(w_lock is true);}

	float
	Rectangle::height()
	const{return	h;}

	const Rectangle &
	Rectangle::rectangle()
	const{return	*this;}

	Rectangle *
	Rectangle::rectangle_address()
	{return	this;}

	Rectangle &
	Rectangle::rectangle_ref()
	{return	*this;}

	float
	Rectangle::area()
	const {return	width()*height();}

	float
	Rectangle::perimeter()
	const{return 2*(w+h);}

	float
	Rectangle::magnitude()
	const{
		float	sx = width()*width();
		float	sy = height()*height();
	return	sqrt(sx+sy);
	}



	bool
	Rectangle::operator	== (const Rectangle & rect){
		if( w isnot rect.w )
			return	false;
	else	if( h isnot rect.h )
			return	false;
	else	if( w_lock isnot rect.w_lock )
			return	false;
	else	if( h_lock isnot rect.h_lock )
			return	false;
		else	return	true;
	}

	bool
	Rectangle::operator	!= (const Rectangle & rect){
		return	( (*this is rect) is false );
	}

	const Rectangle &
	Rectangle::operator	=  (const Rectangle & rect){
		tsx::set(*this,rect);
	return	*this;
	}

	Rectangle
	Rectangle::operator	+ (const Rectangle & rect){
		return	tsx::add(*this,rect);
	}

	const Rectangle &
	Rectangle::operator	+= (const Rectangle & rect){
		return	tsx::add_to(*this,rect);
	}

	Rectangle
	Rectangle::operator	-  (const Rectangle & rect){
		return	tsx::sub(*this,rect);
	}

	const Rectangle &
	Rectangle::operator	-= (const Rectangle & rect){
		return	tsx::sub_from(*this,rect);
	}

	Rectangle
	Rectangle::operator	*  ( float x ){
		Rectangle u(*this);
		tsx::scale(u,x);
	return	u;
	}

	const Rectangle &
	Rectangle::operator	*= ( float x ){
		tsx::scale(*this,x);
	return	*this;
	}






	Point::Point()
	: px(0.0f), py(0.0f), pz(0.0f){
		lx = ly = lz = false;
		auto_lock_set = false;
	}

	Point::Point(float X, float Y, float Z)
	: px(X), py(Y), pz(Z){
		lx = ly = lz = false;
		auto_lock_set = false;
	}

	Point::Point(const Point & A){
		px = A.x();
		py = A.y();
		pz = A.z();

		lx = A.x_locked();
		ly = A.y_locked();
		lz = A.z_locked();

		auto_lock_set = A.auto_lock_set;
	}

	Point::~Point(){}

	const Point &
	Point::point()
	const{return *this;}

	Point *
	Point::point_address()
	{return this;}

	Point &
	Point::point_ref()
	{return *this;}

	// private methods //
	
	bool
	Point::can_set_x()
	const{
		if( point_locked() is true )
			return	false;
	else	if( (ly is true) or (lz is true) )
			return	false;
		else	return	true;
	}

	bool
	Point::can_set_y()
	const{
		if( point_locked() is true )
			return	false;
	else	if( (lx is true) or (lz is true) )
			return	false;
		else	return	true;
	}

	bool
	Point::can_set_z()
	const{
		if( point_locked() is true )
			return	false;
	else	if( (ly is true) or (lx is true) )
			return	false;
		else	return	true;
	}

	// end private //




	// statics and friend zone //

	Point
	Point::create(float a, float b, float c){
		return	Point(a,b,c);
	}

	Point *
	Point::create_address(float a, float b, float c){
		return	new Point(a,b,c);
	}

	Point *
	free_point(Point * at){
		if( at is null )
			return	null;

		delete	at;
	return	(at = null);
	}

	Point *
	Point::free_point(Point * at){
		return	tsx::free_point(at);
	}

	void
	set(Point & at, const Point & place){
		at.x( place.x() );
		at.y( place.y() );
		at.z( place.z() );

		if( auto_locked(at) ){
			tsx::lock_x(at, place.x_locked());
			tsx::lock_y(at, place.y_locked());
			tsx::lock_z(at, place.z_locked());
		}
	}

	void
	Point::set(Point & at, const Point & place){
		tsx::set(at,place);
	}

	void
	Point::set(const Point & place){
		tsx::set(*this,place);
	}

	void
	set(Point & at, float u, float v, float p){
		Point place(u,v,p);
		tsx::set(at,place);
	}

	void
	Point::set(Point & at, float u, float v, float p){
		tsx::set(at,u,v,p);
	}

	void
	Point::set(float u, float v, float p){
		tsx::set(*this,u,v,p);
	}

	Point
	add(const Point & a, const Point & b){
		Point c( a.x() + b.x(), a.y() + b.y(), a.z() + b.z() );

		if( (a.x_locked() is true) or (b.x_locked() is true) )
			c.lock_x();

		if( (a.y_locked() is true) or (b.y_locked() is true) )
			c.lock_y();

		if( (a.z_locked() is true) or (b.z_locked() is true) )
			c.lock_z();

	return	c;
	}

	Point
	Point::add(const Point & a, const Point & b){
		return	tsx::add(a,b);
	}

	Point
	Point::add(const Point & b)
	const{
		return	tsx::add(*this,b);
	}

	Point
	add(const Point & A, float a, float b, float c){
		Point u(A);

		u.x( u.x() + a );
		u.y( u.y() + b );
		u.z( u.z() + c );
	return	u;
	}

	Point
	Point::add(const Point & A, float a, float b, float c){
		return	tsx::add(A,a,b,c);
	}

	Point
	Point::add(float a, float b, float c)
	const{
		return	tsx::add(*this,a,b,c);
	}

	Point
	sub(const Point & A, const Point & B){
		Point	C(A);
		C.x( C.x() - B.x() );
		C.y( C.y() - B.y() );
		C.z( C.z() - B.z() );
	return	C;
	}

	Point
	Point::sub(const Point & A, const Point & B){
		return	tsx::sub(A,B);
	}

	Point
	Point::sub(const Point & A)
	const{
		return	tsx::sub(*this,A);
	}

	const Point &
	sub_from(Point & A, const Point & B){
		A.x( A.x() - B.x() );
		A.y( A.y() - B.y() );
		A.z( A.z() - B.z() );
	return	A;
	}

	const Point &
	Point::sub_from(Point & A, const Point & B){
		return	tsx::sub_from(A,B);
	}

	const Point &
	Point::sub_from(const Point & A){
		return	tsx::sub_from(*this,A);
	}

	const Point &
	add_to(Point & a, const Point & b){
		a.x( a.x() + b.x() );
		a.y( a.y() + b.y() );
		a.z( a.z() + b.z() );
	return	a;
	}

	const Point &
	Point::add_to(Point & a, const Point & b){
		return	tsx::add_to(a,b);
	}

	const Point &
	scale(Point & a, float all){
		a.x( a.x()*all );
		a.y( a.y()*all );
		a.z( a.z()*all );
	return	a;
	}

	const Point &
	Point::scale(Point & a, float all){
		return	tsx::scale(a,all);
	}

	const Point &
	Point::scale(float all){
		return	tsx::scale(*this,all);
	}

	const Point &
	scale(Point & a, float e1, float e2, std::string axiis){
		if( axiis is "xy" ){
			a.x( a.x()*e1 );
			a.y( a.y()*e2 );
	}else	if( axiis is "yz" ){
			a.y( a.y()*e1 );
			a.z( a.z()*e2 );
	}else	if( axiis is "zx" ){
			a.z( a.z()*e1 );
			a.x( a.x()*e2 );
	}else	if( axiis is "yx" ){
			a.y( a.y()*e1 );
			a.x( a.x()*e2 );
	}else	if( axiis is "zy" ){
			a.z( a.z()*e1 );
			a.y( a.y()*e2 );
	}else	if( axiis is "xz" ){
			a.x( a.x()*e1 );
			a.z( a.z()*e2 );
	}

	return	a;
	}

	const Point &
	Point::scale(Point & a, float e1, float e2, std::string axiis){
		return	tsx::scale(a,e1,e2,axiis);
	}

	const Point &
	Point::scale(float a, float b, std::string aa){
		return	tsx::scale(*this,a,b,aa);
	}

	const Point &
	scale(Point & a, float w, float u, float v){
		a.x( a.x()*w );
		a.y( a.y()*u );
		a.z( a.z()*v );
	return	a;
	}

	const Point &
	Point::scale(Point & a, float w, float u, float v){
		return	tsx::scale(a,w,u,v);
	}

	const Point &
	Point::scale(float w, float u, float v){
		return	tsx::scale(*this,w,u,v);
	}

	const Point &
	scale(Point & a, const Point & b){
		return	tsx::scale(a,b.x(), b.y(), b.z());
	}

	const Point &
	Point::scale(Point & a, const Point & b){
		return	tsx::scale(a,b);
	}

	const Point &
	Point::scale(const Point & b){
		return	tsx::scale(*this,b);
	}

	float
	product(const Point & a, const Point & b){
		float sum = 0.0f;
		sum += a.x() * b.x();
		sum += a.y() * b.y();
		sum += a.z() * b.z();
	return	sum;
	}

	float
	Point::product(const Point & a, const Point & b){
		return	tsx::product(a,b);
	}

	float
	Point::product(const Point & a)
	const{
		return	tsx::product(*this,a);
	}

	float
	distance(const Point & A, const Point & B){
		float	dx = A.x() - B.x();
		float	dy = A.y() - B.y();
		float	dz = A.z() - B.z();

		float	sx = dx*dx;
		float	sy = dy*dy;
		float	sz = dz*dz;
	return	sqrtf(sx+sy+sz);
	}

	float
	Point::distance(const Point & A, const Point & B){
		return	tsx::distance(A,B);
	}

	float
	Point::distance(const Point & A)
	const{
		return	distance(*this,A);
	}

	float
	distance(const Point & A, float u, float v, float w){
		float	dx = A.x() - u;
		float	dy = A.y() - v;
		float	dz = A.z() - w;

		float	sx = dx*dx;
		float	sy = dy*dy;
		float	sz = dz*dz;
	return	sqrtf(sx+sy+sz);
	}

	float
	Point::distance(const Point & A, float u, float v, float w){
		return	tsx::distance(A,u,v,w);
	}

	float
	Point::distance(float u, float v, float w)
	const{
		return	tsx::distance(*this,u,v,w);
	}

	float
	magnitude(const Point & A){
		return	tsx::distance(A,0.0f,0.0f,0.0f);
	}

	float
	Point::magnitude(const Point & A){
		return	tsx::magnitude(A);
	}

	float
	Point::magnitude()
	const{return	magnitude(*this);}
	
	// end friends and statics //

	float
	Point::x()
	const{return px;}

	void
	Point::x(float nx){
		if( can_set_x() is true )
			px = nx;
	}

	void
	Point::y(float ny){
		if( can_set_y() is true )
			py = ny;
	}

	void
	Point::z(float nz){
		if( can_set_z() is true )
			pz = nz;
	}

	float
	Point::y()
	const{return py;}

	float
	Point::z()
	const{return pz;}

	bool
	Point::x_locked()
	const{return lx;}

	bool
	Point::x_locked(const Point & at){
		return	at.x_locked();
	}

	bool
	x_locked(const Point & at){
		return	at.x_locked();
	}

	bool
	Point::y_locked()
	const{return ly;}

	bool
	Point::y_locked(const Point & at){
		return at.y_locked();
	}

	bool
	y_locked(const Point & at){
		return	at.y_locked();
	}

	bool
	Point::z_locked()
	const{return lz;}

	bool
	Point::z_locked(const Point & at){
		return	at.z_locked();
	}

	bool
	z_locked(const Point & at){
		return	at.z_locked();
	}


	void
	Point::lock_x(bool lock)
	{lx=lock;}

	void
	lock_x(Point & x, bool lock){
		x.lock_x(lock);
	}

	void
	Point::lock_x(Point & at, bool l){
		return	tsx::lock_x(at,l);
	}

	void
	lock_y(Point & at, bool l){
		return	at.lock_y(l);
	}

	void
	Point::lock_y(Point & at, bool l){
		return	tsx::lock_y(at,l);
	}

	void
	Point::lock_y(bool lock)
	{ly=lock;}

	void
	Point::lock_z(bool lock)
	{lz=lock;}

	void
	lock_z(Point & at, bool lock){
		at.lock_z(lock);
	}
	
	void
	Point::lock_z(Point & at, bool l){
		tsx::lock_z(at,l);
	}

	void
	remove_locks(Point & at){
		at.remove_locks();
	}

	void
	Point::remove_locks(Point & at){
		tsx::remove_locks(at);
	}

	void
	Point::remove_locks(){
		lx = ly = lz = false;
	}

	bool
	Point::point_locked()
	const{
		return( (lx is true) or (ly is true) or (lz is true) );
	}

	bool
	point_locked(const Point & at){
		return	at.point_locked();
	}

	bool
	Point::point_locked(const Point & at){
		return	tsx::point_locked(at);
	}

	bool
	Point::auto_locked()
	const{
		return	(auto_lock_set is true);
	}

	bool
	auto_locked(const Point & at){
		return	at.auto_locked();
	}

	bool
	Point::auto_locked(const Point & at){
		return	tsx::auto_locked(at);
	}

	void
	Point::auto_lock(bool v){
		auto_lock_set = v;
	}

	void
	auto_lock(Point & at, bool v){
		at.auto_lock(v);
	}

	void
	Point::auto_lock(Point & at, bool v){
		tsx::auto_lock(at,v);
	}



	bool
	Point::operator	== (const Point & at){
		if( x() isnot at.x() )
			return	false;
	else	if( y() isnot at.y() )
			return	false;
	else	if( z() isnot at.z() )
			return	false;
	else	if( x_locked() isnot x_locked(at) )
			return	false;
	else	if( y_locked() isnot y_locked(at) )
			return	false;
	else	if( z_locked() isnot z_locked(at) )
			return	false;
		else	return	true;
	}

	bool
	Point::operator != (const Point & at){
		return	( (*this is at) isnot true );
	}

	const Point &
	Point::operator	 = (const Point & at){
		tsx::set(*this,at);
	return	*this;
	}

	Point
	Point::operator	 + (const Point & at){
		return	tsx::add(*this,at);
	}

	const Point &
	Point::operator	+= (const Point & at){
		return	tsx::add_to(*this,at);
	}

	Point
	Point::operator	 - (const Point & at){
		return	tsx::sub(*this,at);
	}

	const Point &
	Point::operator	-= (const Point & at){
		return	tsx::sub_from(*this,at);
	}

	Point
	Point::operator	 * (const Point & at){
		Point place(*this);
		tsx::scale(place,at);
	return	place;
	}

	Point
	Point::operator	 * (float x){
		Point place(*this);
		tsx::scale(place,x);
	return	place;
	}

	const Point &
	Point::operator *= (float x){
		return	tsx::scale(*this,x);
	}

	const Point &
	Point::operator	*= (const Point & at){
		return	tsx::scale(*this,at);
	}




}



